#include "hip/hip_runtime.h"
#include "common.cpp"
#include <hip/hip_runtime_api.h>
#include <float.h>
#include ""
#include "kernels.cu"
# define cudaCheck\
 {\
 hipError_t err = hipGetLastError ();\
 if ( err != hipSuccess ){\
 printf(" hipError_t = '%s' \n in '%s' %d\n", hipGetErrorString( err ), __FILE__ , __LINE__ );\
 exit(0);}}


void classifier(svm_model *model, svm_sample *test, float *rate)
{
	float reductiontime = 0;
	float intervaltime;
	hipEvent_t start, stop;
	hipEventCreate ( &start );cudaCheck
	hipEventCreate ( &stop  );cudaCheck

	int nTV = test->nTV;
	int nSV = model->nSV;
	int nfeatures = model->nfeatures;

	float *d_TV = 0;	
	float *d_SV = 0;
	hipMalloc((void**) &d_SV, nSV*nfeatures*sizeof(float));cudaCheck
	hipMemcpy(d_SV, model->SV_dens, nSV*nfeatures*sizeof(float),hipMemcpyHostToDevice);cudaCheck

		float *d_l_SV = 0;
	hipMalloc((void**) &d_l_SV, nSV*sizeof(float));cudaCheck
	hipMemcpy(d_l_SV, model->l_SV, nSV*sizeof(float),hipMemcpyHostToDevice);cudaCheck

	size_t remainingMemory = 0;
	size_t totalMemory = 0;
	hipMemGetInfo(&remainingMemory, &totalMemory);	cudaCheck
	int cache_size = remainingMemory/(nSV*sizeof(float)); // # of TVs in cache
	if (nTV <= cache_size){	cache_size = nTV; }

	hipMalloc((void**) &d_TV, cache_size*nfeatures*sizeof(float));cudaCheck

	int nthreads = MAXTHREADS;
	int nblocks_cache = min(MAXBLOCKS, (cache_size + nthreads - 1)/nthreads);
	int nblocks_SV = min(MAXBLOCKS, (nSV + nthreads - 1)/nthreads);
	dim3 dim_block = dim3(nblocks_cache, 1, 1);
	dim3 dim_thread = dim3(MAXTHREADS, 1, 1);
	// Allocate device memory for F
	float* h_fdata= (float*) malloc(nblocks_SV*sizeof(float));
	float* d_fdata=0;
	hipMalloc((void**) &d_fdata, nblocks_SV*sizeof(float));cudaCheck
	int offset = 0;
	int num_of_parts =  (nTV + cache_size - 1)/cache_size;
	int* h_l_estimated = (int*)malloc(test->nTV*sizeof(int));
	for (int ipart = 0; ipart < num_of_parts; ipart++)
	{
		if ((ipart == (num_of_parts - 1)) && ((nTV - offset) != 0) )
		{
			cache_size = nTV - offset;
		}
		hipMemcpy(d_TV, &test->TV[offset*nfeatures], cache_size*nfeatures*sizeof(float),hipMemcpyHostToDevice);cudaCheck
			for (int i = 0; i < cache_size; i++)
			{				
				reduction<<<nblocks_SV, MAXTHREADS, MAXTHREADS*sizeof(float)>>>(d_SV, &d_TV[i*nfeatures], d_l_SV, nSV, nfeatures, model->coef_gamma, model->kernel_type, d_fdata);cudaCheck
				hipMemcpy(h_fdata, d_fdata, nblocks_SV*sizeof(float), hipMemcpyDeviceToHost); cudaCheck

				float sum = 0;
				for (int k = 0; k < nblocks_SV; k++)
					sum += h_fdata[k];

				sum -= model->b;
				if (sum > 0)
				{
					h_l_estimated[i + offset] = 1;
				}
				else
				{
					h_l_estimated[i + offset] = -1;
				}
			}
			offset += cache_size;
	}
	hipFree(d_fdata);cudaCheck
	hipFree(d_l_SV);cudaCheck
	hipFree(d_SV);cudaCheck
	hipFree(d_TV);cudaCheck
	hipDeviceReset();cudaCheck

	int errors=0;
	for (int i=0; i<test->nTV; i++)
	{
		if( test->l_TV[i]!=h_l_estimated[i])
		{
			errors++;
		}
	}
	*rate = (float)(test->nTV - errors)/test->nTV;
	
	free(h_l_estimated);
	free(h_fdata);
}
void Reduce_step(int *d_y, float *d_a, float *d_f, float *d_B, unsigned int *d_I, float *param, int ntraining, int nblocks,
				 float *h_B, unsigned int *h_I, float* h_B_global, unsigned int *h_I_global, int *active, int ntasks)
{
	int smem = MAXTHREADS*(sizeof(float) + sizeof(int));
	for (int itask = 0; itask < ntasks; itask++)
	{
		if (active[itask] == 1)
		{
			Local_Reduce_Min<<<nblocks, MAXTHREADS, smem>>>(d_y, &d_a[itask*ntraining], &d_f[itask*ntraining], &d_B[itask*2*nblocks], &d_I[itask*2*nblocks], param[2*itask], ntraining);
			Local_Reduce_Max<<<nblocks, MAXTHREADS, smem>>>(d_y, &d_a[itask*ntraining], &d_f[itask*ntraining], &d_B[itask*2*nblocks+nblocks], &d_I[itask*2*nblocks+nblocks], param[2*itask], ntraining);
		}
	}
	hipMemcpy(h_B, d_B, ntasks*2*nblocks*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_I, d_I, ntasks*2*nblocks*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int itask = 0; itask < ntasks; itask++)
	{
		if (active[itask] == 1)
		{
			// Global reduction
			float global_Bup = h_B[itask*2*nblocks];
			float global_Blow = h_B[itask*2*nblocks+nblocks];
			int global_Iup = h_I[itask*2*nblocks];
			int global_Ilow = h_I[itask*2*nblocks+nblocks];

			for (int i = 1; i < nblocks; i++)
			{
				if (h_B[itask*2*nblocks+i] < global_Bup)
				{
					global_Bup = h_B[itask*2*nblocks+i];
					global_Iup = h_I[itask*2*nblocks+i];
				}
				if (h_B[itask*2*nblocks+nblocks + i] > global_Blow)
				{
					global_Blow = h_B[itask*2*nblocks+nblocks + i];
					global_Ilow = h_I[itask*2*nblocks+nblocks + i];
				}
			}

			h_B_global[itask*2] = global_Bup;
			h_B_global[itask*2+1] = global_Blow;
			h_I_global[itask*2] = global_Iup;
			h_I_global[itask*2+1] = global_Ilow;
		}
	}
}


void cross_validation(svm_sample *train, svm_model *model)
{
	hipEvent_t start, stop;
	hipEventCreate ( &start );
	hipEventCreate ( &stop  );

	int ntasks = model->ntasks;
	int nTV = train->nTV;
	int nfeatures = model->nfeatures;
	//Grid configuration
	int nthreads = MAXTHREADS;
	int nblocks = min(MAXBLOCKS, (nthreads + nTV - 1)/nthreads);

	float *d_TV = 0;//training vectors
	hipMalloc((void**) &d_TV, nTV*nfeatures*sizeof(float));
	hipMemcpy(d_TV, train->TV, nTV*nfeatures*sizeof(float),hipMemcpyHostToDevice);

	float *d_SV = 0;//support vectors

	float *d_params = 0;// binary labels
	hipMalloc((void**) &d_params, 2*ntasks*sizeof(float));
	hipMemcpy(d_params, model->params, 2*ntasks*sizeof(float),hipMemcpyHostToDevice);

	int *d_y = 0;// binary labels
	hipMalloc((void**) &d_y, nTV*sizeof(int));
	hipMemcpy(d_y, train->l_TV, nTV*sizeof(int),hipMemcpyHostToDevice);

	float *d_a = 0; //alphas
	hipMalloc((void**) &d_a, ntasks*nTV*sizeof(float));

	float *h_f = (float*)malloc(ntasks*nTV*sizeof(float));
	float *d_f = 0;//object functions
	hipMalloc((void**) &d_f, ntasks*nTV*sizeof(float));

	//locally reduced thresholds {Bup:Blow}
	float *h_B = (float*)malloc(2*nblocks*ntasks*sizeof(float));
	float *d_B = 0;
	hipMalloc((void**) &d_B, 2*nblocks*ntasks*sizeof(float));

	//indeces of locally reduced Lagrange multipliers {Iup:Ilow}
	unsigned int *h_I = (unsigned int*)malloc(2*nblocks*ntasks*sizeof(unsigned int));
	unsigned int *d_I = 0; 
	hipMalloc((void**) &d_I, 2*nblocks*ntasks*sizeof(unsigned int));

	//global tresholds {Bup:Blow}
	float *h_B_global = (float*)malloc(2*ntasks*sizeof(float));

	unsigned int *h_I_global = (unsigned int*)malloc(2*ntasks*sizeof(unsigned int));
	unsigned int *d_I_global = 0; 
	hipMalloc((void**) &d_I_global, 2*ntasks*sizeof(unsigned int));
	unsigned int *h_I_cache = (unsigned int*)malloc(2*ntasks*sizeof(unsigned int));
	unsigned int *d_I_cache = 0; 
	hipMalloc((void**) &d_I_cache, 2*ntasks*sizeof(unsigned int));

	float *h_delta_a = (float*)malloc(2*ntasks*sizeof(float));
	float *d_delta_a = 0;
	hipMalloc((void**) &d_delta_a, 2*ntasks*sizeof(float));

	int *h_active = (int*)malloc(ntasks*sizeof(int));
	for (int i = 0; i < ntasks; i++)
		h_active[i] = 1;

	int *d_active = 0;
	hipMalloc((void**) &d_active, ntasks*sizeof(int));
	hipMemcpy(d_active, h_active, ntasks*sizeof(int),hipMemcpyHostToDevice);

	initialization<<<dim3(nblocks, 1), dim3(nthreads, ntasks)>>>(d_a, d_f, d_y, nTV);
	Reduce_step(d_y, d_a, d_f, d_B, d_I, model->params, nTV, nblocks, h_B, h_I, h_B_global, h_I_global, h_active, ntasks);

	unsigned int remainingMemory;
	unsigned int totalMemory;
	hipMemGetInfo(&remainingMemory, &totalMemory);

	printf("%u bytes of memory found on device, %u bytes currently free\n", totalMemory, remainingMemory);

	int sizeOfCache = remainingMemory/(nTV*sizeof(float));

	sizeOfCache = (int)((float)sizeOfCache*KMEM);
	if (nTV < sizeOfCache)
		sizeOfCache = nTV;

	printf("%u rows of kernel matrix will be cached (%u bytes per row)\n", sizeOfCache, nTV*sizeof(float));

	float *d_k = 0;// gramm matrix
	hipMalloc((void**) &d_k, sizeOfCache*nTV*sizeof(float));

	hipStream_t *stream = (hipStream_t*)malloc(2*sizeof(hipStream_t));
	for (int i = 0; i < 2; i++)
	{
		hipStreamCreate(&stream[i]);
	}

	int iter = 0;
	std::list<std::pair<unsigned int, unsigned int>>cache;

	while (chech_condition(h_B_global, h_active, ntasks))
	{
		++iter;	
		for (int itask = 0; itask < ntasks; itask++)
		{
			if (h_active[itask] == 1)
			{
				if(check_cache(h_I_global[2*itask], &h_I_cache[2*itask], &cache, sizeOfCache))		//Iup - second
					get_row<<<nblocks, nthreads,0,stream[0]>>>(d_k, d_TV, model->params[2*itask+1], nfeatures, h_I_global[2*itask], h_I_cache[2*itask], nTV);
				if(check_cache(h_I_global[2*itask+1], &h_I_cache[2*itask+1], &cache, sizeOfCache))//Ilow - fist
					get_row<<<nblocks, nthreads,0,stream[1]>>>(d_k, d_TV, model->params[2*itask+1], nfeatures, h_I_global[2*itask+1], h_I_cache[2*itask+1], nTV);
			}
		}

		hipMemcpy(d_active, h_active, ntasks*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_I_cache, h_I_cache, ntasks*2*sizeof(unsigned int),hipMemcpyHostToDevice);
		hipMemcpy(d_I_global, h_I_global, ntasks*2*sizeof(unsigned int),hipMemcpyHostToDevice);

		Update<<<1,ntasks>>>(d_k, d_y, d_f, d_a, d_delta_a, d_I_global, d_I_cache, d_params, d_active, nTV);
		hipMemcpy(h_delta_a, d_delta_a, 2*ntasks*sizeof(float),hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		Map<<<dim3(nblocks, 1), dim3(nthreads, ntasks)>>>(d_f, d_k, d_y, d_delta_a, d_I_global, d_I_cache, d_active, nTV);
		hipDeviceSynchronize();
		hipMemcpy(h_f, d_f, nTV*ntasks*sizeof(float), hipMemcpyDeviceToHost);
		Reduce_step(d_y, d_a, d_f, d_B, d_I, model->params, nTV, nblocks, h_B, h_I, h_B_global, h_I_global, h_active, ntasks);
	}
	printf("All tasks convergented in %f\n", cuGetTimer());

	//predict
	model->l_SV = (float*)malloc(nTV*ntasks*sizeof(float));
	hipMemcpy(model->l_SV, d_a, nTV*ntasks*sizeof(float), hipMemcpyDeviceToHost);
	model->mass_b = (float*)malloc(ntasks*sizeof(float));
	for (int itask = 0; itask < model->ntasks; itask++)
	{
		model->mass_b[itask] = (h_B_global[2*itask+1]+h_B_global[2*itask])/2;
	}
	hipDeviceReset();
}

void classification( svm_sample *test, svm_model *model)
{
	int nTV = test->nTV;
	float *buf_l = model->l_SV;
	float rate;
	float max_rate = 0;
	int max_rate_ind;

	for (int itask = 0; itask < model->ntasks; itask++)
	{
		float *SV = (float*)malloc(test->nTV*model->nfeatures*sizeof(float));
		float *l_sv = (float*)malloc(nTV*sizeof(float));
		model->l_SV = &buf_l[itask*nTV];
		int nSV = 0;
		for (int i = 0; i < nTV; i++)
		{
			if (model->l_SV[i] != 0)
			{
				if (i != nSV)
				{
					l_sv[nSV] = test->l_TV[i]*model->l_SV[i];
					for (int j = 0; j < model->nfeatures; j++)
						SV[nSV*model->nfeatures+j] = test->TV[i*model->nfeatures+j];
				}	
				++nSV;
			}
		}
		model->nSV = nSV;
		model->b = model->mass_b[itask];
		model->C = model->params[2*itask];
		model->coef_gamma = model->params[2*itask+1];
		model->l_SV=(float*)realloc(l_sv, nSV*sizeof(float));
		model->SV_dens=(float*)realloc(SV, nSV*model->nfeatures*sizeof(float));
		classifier(model, test, &rate);
		if (max_rate < rate)
		{
			max_rate = rate;
			max_rate_ind = itask;
		}
		free(model->l_SV);
		free(model->SV_dens);
		printf("Task %d occuracy is %f with C=%f and gamma=%f #SV=%d\n",itask, rate, model->params[2*itask], model->params[2*itask+1], nSV);
	}
	printf("best occuracy is %f with C=%f and gamma=%f\n", max_rate, model->params[2*max_rate_ind], model->params[2*max_rate_ind+1]);

	//free(model->label_set);
	//free(model->mass_b);
	//free(model->params);
	//free(model);
	//free(test->l_TV);
	//free(test->TV);
	//free(test);
}

int main(int argc, char **argv)
{
	FILE *input, *output;
	if (argc==1)
	{
		argc = 4;
		//argv[1] = "C:\\Data\\b.txt";
		//argv[2] = "C:\\Data\\b.model";
		//argv[3] = "10";
		argv[1] = "C:\\Data\\a9a";
		argv[2] = "C:\\Data\\a9a.model";
		argv[3] = "123";
		//argv[1] = "C:\\Data\\mushrooms";
		//argv[2] = "C:\\Data\\mushrooms.model";
		//argv[3] = "112";
		//argv[1] = "C:\\Data\\ijcnn1";
		//argv[2] = "C:\\Data\\ijcnn1.model";
		//argv[3] = "22";

	}
	if(argc<4)
		exit_with_help();
	struct svm_model *model = (svm_model*)malloc(sizeof(svm_model));
	struct svm_sample *train = (svm_sample*)malloc(sizeof(svm_sample));
	struct svm_sample *test = (svm_sample*)malloc(sizeof(svm_sample));
	sscanf(argv[3],"%d",&model->nfeatures);

	if((input = fopen(argv[1],"r")) == NULL)
	{
		fprintf(stderr,"can't open training file %s\n",argv[1]);
		exit(1);
	}

	if((output = fopen(argv[2],"w")) == NULL)
	{
		fprintf(stderr,"can't create model file %s\n",argv[2]);
		exit(1);
	}
	float percent = 0.8;
	set_model_param(model, 1, 4, 0.01, 2);
	converg_time= (float*)malloc(model->ntasks*sizeof(float));
	for (int itask = 0; itask < model->ntasks; itask++)
		converg_time[itask] = 0;
	
	parse_TV(input, train, model);
	set_labels(train, model);
	balabce_data(train, test, percent);
	cuResetTimer();
	cross_validation(train, model);
	for (int itask = 0; itask < model->ntasks; itask++)
		printf("Task %d has convergent in %f\n", itask, converg_time[itask]);
	classification(test, model);
	printf("Total time %f cache hits %d\n", cuGetTimer(), cache_hit);
	hipDeviceReset();
	return 0;
}
